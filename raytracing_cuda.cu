#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <float.h>
#include <time.h>
const int width = 2048;
const int height = width;
const int maxReflect = 5;
struct Color {
  unsigned char b, g, r, a;
};

Color* image;
void writebmpheader(FILE* f, int width, int height) {
  int size = width * height * sizeof(Color);
  struct {
    uint32_t filesz;
    uint16_t creator1;
    uint16_t creator2;
    uint32_t bmp_offset;
  } bmpheader = 
    { size + 54, 0, 0, 54};
  struct {
    uint32_t header_sz;
    int32_t width;
    int32_t height;
    uint16_t nplanes;
    uint16_t bitspp;
    uint32_t compress_type;
    uint32_t bmp_bytesz;
    int32_t hres;
    int32_t vres;
    uint32_t ncolors;
    uint32_t nimpcolors;    
  } dibheader = 
    {40, width, height, 1, 32, 0, size, 0, 0, 0, 0};
  fwrite("BM", 2, 1, f);
  fwrite(&bmpheader, sizeof(bmpheader), 1, f);
  fwrite(&dibheader, sizeof(dibheader), 1, f);
}
void writebmp(const char* filename, const Color* data, int width, int
              height) {
  FILE* f = fopen(filename, "wb");
  if (!f) return;
  writebmpheader(f, width, height);
  fwrite(data, sizeof(Color), width * height, f);
  fclose(f);
}

__device__ __host__ inline float3 operator+(float3 a, float3 b) {
  return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__device__ __host__ inline float3 operator-(float3 a, float3 b) {
  return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__device__ inline float3& operator+=(float3& a, float3 b) {
  a.x += b.x;
  a.y += b.y;
  a.z += b.z;
  return a;
}
__device__ __host__ inline float3 cross(float3 a, float3 b) {
  return make_float3( -a.z * b.y + a.y * b.z,
                      a.z * b.x - a.x * b.z,
                      -a.y * b.x + a.x * b.y );
}
__device__ inline float dot(float3 a, float3 b) {
  return a.x * b.x + a.y * b.y + a.z * b.z;
}
__device__ __host__ float3 inline operator*(float3 a, float b) {
  return make_float3(a.x * b, a.y * b, a.z * b);
}
__device__ __host__ float3 inline operator*(float b, float3 a) {
  return a * b;
}

__device__ __host__ inline float sqrlength(float3 v) {
  return v.x * v.x + v.y * v.y + v.z * v.z;
}
__device__ __host__ inline float length(float3 v) {
  return sqrtf(sqrlength(v));
}
__device__ __host__ inline float3 normalize(float3 v) {
  float invlen = 1 / length(v);
  return make_float3(v.x * invlen, v.y * invlen, v.z * invlen);
}
__device__ inline float3 modulate(float3 a, float3 b) {
  return make_float3(a.x * b.x, a.y * b.y, a.z * b.z);
}
struct Ray {
  float3 origin, direction;

  __device__ float3 getPoint(float t) {
    return origin + t * direction;
  }
};

struct PerspectiveCamera {
  float3 eye, front, right, up;
  float fovScale;

  __device__ Ray generateRay(float x, float y) const {
    float3 r = right* ((x - 0.5) * fovScale);
    float3 u = up * ((y - 0.5) * fovScale);
    Ray ray = {eye, normalize(front + r + u)};
    return ray;
  }
};

PerspectiveCamera makePerspectiveCamera(float3 e, float3 f, float3 u, float v) {
  PerspectiveCamera c = {e, f, cross(f, u), cross(cross(f,u), f) , tan(v * 0.5 * 3.1415926 / 180) * 2};
  return c;
};
enum g_t {G_SPHERE, G_PLANE} ;
struct IntersectResult {
  g_t g_type;
  int g_id;
  float distance, reflectiveness;
  float3 position, normal;
};

struct Sphere {
  float3 center;
  float radius, sqrRadius;
  float3 diffuse, specular;
  int shininess;
  float reflectiveness;
  float3 lightDir;
  float3 lightColor;

  __device__ inline bool intersect(Ray& ray, IntersectResult& result) const {
    float3 v = ray.origin - center;
    float a0 = sqrlength(v) - sqrRadius;
    float DdotV = dot(ray.direction, v);
    if (DdotV <= 0) {
      float discr = DdotV * DdotV - a0;
      if (discr >= 0) {
        result.g_type = G_SPHERE;
        result.distance = -DdotV - sqrt(discr);
        result.position = ray.getPoint(result.distance);
        result.normal = normalize(result.position - center);
        result.reflectiveness = reflectiveness;
        return true;
      }
    }
    return false;
  }

  __device__ inline float3 sample(Ray ray, float3 position, float3 normal) const {
    
    float NdotL = dot(normal, lightDir);
    float3 H = normalize(lightDir - ray.direction);
    float NdotH = dot(normal, H);
    float3 diffuseTerm = diffuse * fmaxf(NdotL, 0.0);
    float3 specularTerm = specular * __powf(fmaxf(NdotH, 0.0), shininess);
    return modulate(lightColor, diffuseTerm + specularTerm);
    
  }
};

Sphere makeSphere(float3 c, float r, float3 d, float3 sp, int sh, float re = 0.0) {
  Sphere s = {
    c, r, r * r, d, sp, sh ,re,
    normalize(make_float3(1, 1, 1)),
    make_float3(1, 1, 1)
  };
  return s;
};
struct Plane {
  float3 normal, position;
  float scale, reflectiveness;
  __device__ inline bool intersect(Ray ray, IntersectResult& result) const {
    float a = dot(ray.direction, normal);
    if (a >= 0.0)
      return false;
    float b = dot(normal, ray.origin - position);
    float d = -b / a;
    result.g_type = G_PLANE;
    result.distance = d;
    result.position = ray.getPoint(d);
    result.normal = normal;
    result.reflectiveness = reflectiveness;
    return true;
  }

  __device__ inline float3 sample(Ray ray, float3 position, float3 normal) const {
    if (fmodf(fabsf(floorf(position.x * 0.1) + floorf(position.z * scale)), 2) < 1)
      return make_float3(0, 0, 0);
    else
      return make_float3(1, 1, 1);
  }
};

Plane makePlane(float3 n, float d, float s, float r) {
  Plane p = {n, n * d, s, r};
  return p;
};
struct RayTracingParam {
  PerspectiveCamera camera;
  int spheres_n;
  Sphere spheres[10];
  int planes_n;
  Plane planes[10];
  int maxReflect;
} cpuparam = 
{
  makePerspectiveCamera(make_float3(0, 5, 15), make_float3(0, 0, -1),
                    make_float3(0, 1, 0), 90),
  2,
  {makeSphere(make_float3(-10, 10, -10), 10, 
          make_float3(1, 0, 0), make_float3(1, 1, 1), 16, 0.25),
   makeSphere(make_float3(10, 10, -10), 10, 
          make_float3(0, 0, 1), make_float3(1, 1, 1), 16, 0.25)},
   1,
   {makePlane(make_float3(0, 1, 0), 0, 0.1, 0.25)}
};

__constant__ RayTracingParam param;
template <typename T>
__device__ inline bool intersect(T* geometries, int n, Ray r, IntersectResult& result) {
  IntersectResult ir;
  bool ok = false;
  for (int i = 0; i < n; ++i) {
    ir.g_id = i;
    if (geometries[i].intersect(r, ir) && ir.distance < result.distance) {
      result.distance = ir.distance;
      result = ir;
      ok = true;
    }
  }
  return ok;
}

__device__ inline bool intersect(Ray r, IntersectResult& result) {
  bool ok = false;
  result.distance = FLT_MAX;
  ok = intersect(param.spheres, param.spheres_n, r, result) || ok;
  ok = intersect(param.planes, param.planes_n, r, result) || ok;
  return ok;
}

__device__ inline float3 sample(Ray r, int g_type, int g_id, float3 position,
                                float3 normal) {
  if (g_type)
    return param.planes[g_id].sample(r, position, normal);
  else 
    return param.spheres[g_id].sample(r, position, normal);
}

__device__ inline float3 gpuSample(Ray ray) {
  float3 color = make_float3(0, 0, 0);
  float reflectiveness = 1.0;
  float r = 1.0;
  float3 c = make_float3(0, 0, 0);
  IntersectResult ir;
  
  for (int i = 0; i < maxReflect + 1; ++i) {
    if (!intersect(ray, ir)) break;
    color += reflectiveness * (1 - r) * c;
    reflectiveness = reflectiveness * r;
    r = ir.reflectiveness;
    c = sample(ray, ir.g_type, ir.g_id, ir.position, ir.normal);
    if (r > 0) {
      Ray newray = {ir.position,
                 ir.normal * (-2*dot(ir.normal,ray.direction)) + ray.direction};
      ray = newray;
    } else
      break;
  }
  return color + reflectiveness * c;
}

__global__ void gpuRayTracing(unsigned* out) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  
  int y = index / width, x = index % width;
  float sx = x / float(width), sy = y / float(height);
  Ray r = param.camera.generateRay(sx, sy);
  float3 c = gpuSample(r);
  unsigned char c4[] = {
      __saturatef(c.z) * 255,
      __saturatef(c.y) * 255,
      __saturatef(c.x) * 255,
      255};

  unsigned ct = *reinterpret_cast<unsigned*>(c4); 
  out[index] = ct;
}

int main() {
  unsigned* gpuout;
  hipSetDevice(0);
  hipHostMalloc(&image, width * height * sizeof(Color));
  hipMalloc(&gpuout, sizeof(Color) * width * height);
  hipMemcpyToSymbol(HIP_SYMBOL(param), &cpuparam, sizeof RayTracingParam);
  clock_t t1 = clock(); 
  gpuRayTracing<<<width * height / 256, 256>>>(gpuout);
  hipMemcpy(image, gpuout, sizeof(Color) * width * height, hipMemcpyDeviceToHost); 
  clock_t t2 = clock();

  printf("%f\n", (t2 - t1) / float(CLOCKS_PER_SEC));
  hipFree(gpuout);
  writebmp("raytracing_cuda.bmp", image, width, height);
  hipHostFree(image);
  return 0;
}
